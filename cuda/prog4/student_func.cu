#include "hip/hip_runtime.h"
//Udacity HW 4
//Radix Sorting

#include <stdio.h>
#include "reference_calc.cpp"
#include "utils.h"

/* Red Eye Removal
===============

For this assignment we are implementing red eye removal.  This is
accomplished by first creating a score for every pixel that tells us how
likely it is to be a red eye pixel.  We have already done this for you - you
are receiving the scores and need to sort them in ascending order so that we
know which pixels to alter to remove the red eye.

Note: ascending order == smallest to largest

Each score is associated with a position, when you sort the scores, you must
also move the positions accordingly.

Implementing Parallel Radix Sort with CUDA
==========================================

The basic idea is to construct a histogram on each pass of how many of each
"digit" there are.   Then we scan this histogram so that we know where to put
the output of each digit.  For example, the first 1 must come after all the
0s so we have to know how many 0s there are to be able to start moving 1s
into the correct position.

1) Histogram of the number of occurrences of each digit
2) Exclusive Prefix Sum of Histogram
3) Determine relative offset of each digit
For example [0 0 1 1 0 0 1]
->  [0 1 0 1 2 3 2]
4) Combine the results of steps 2 & 3 to determine the final
output location for each element and move it there

LSB Radix sort is an out-of-place sort and you will need to ping-pong values
between the input and output buffers we have provided.  Make sure the final
sorted results end up in the output buffer!  Hint: You may need to do a copy
at the end.


Radix sort description
1. Start with LSB
2. Split input into 2 sets based on bit. OTW preserve order
3. Move to next MSB, repeat

*/

/*
    TODO Prefix scan
    Credit goes to Mark Harris at NVIDIA
*/
unsigned int __device__ plus_scan(unsigned int *x){
  unsigned int i = threadIdx.x;
  unsigned int n = blockDim.x;
  unsigned int offset;
  unsigned int y;

  for(offset = 1; offset < n; offset *= 2){
    if (i >= offset)
      y = x[i-offset];

    __syncthreads();

    if (i >= offset)
      x[i] = y + x[i];
    __syncthreads();
  }
  return x[i];
}

/*
    Patition s.t. all values with a 0 at the bit index preceed those with a 1
    Heavily inspired from Mark Harris' example functions provided in the course
    materials
*/
__device__ void partition_by_bit(unsigned int* d_in, unsigned int bit){
  unsigned int i = threadIdx.x;
  unsigned int size = blockDim.x;
  // value at position i
  unsigned int x_i = d_in[i];
  // mask to get binary value at index bit
  unsigned int p_i = (x_i >> bit) & 1;

  // replace real value with binary value
  d_in[i] = p_i;
  __syncthreads();

  // compute number of 1's and update d_in s.t. it contains the sum of the 1's
  // from d[0] .. d[i]
  // TODO
  unsigned int before = plus_scan(d_in);


  // barrier in the plus_scan function means that we are synced at this point

  unsigned int o_total = d_in[size-1]; // number of ones in array
  unsigned int z_total = size - o_total; // number of zeros

  __syncthreads();

  // rearrage the values. This is a permutation of the array
  if (p_i)
    d_in[o_total-1 + z_total] = x_i;
  else
    d_in[i - before] = x_i;

}

/*
Kernel function. Put value into appropriate bin.
*/
__global__ void bin_hist(unsigned int * d_bins, unsigned int* d_in, int size,
  int numBins){
    unsigned int b; // bit

    // partition by bit
    for (b = 0; b < 8 * sizeof(unsigned int); b+=2 ){

      // mask off a set number of bits each step for AND operation
      unsigned int mask = (numBins - 1) << b;

      int index = threadIdx.x + blockIdx.x * blockDim.x;

      unsigned int bin = (d_in[index] & mask) >> b;

      atomicAdd(&(d_bins[bin]), 1);

      // keep threads in lock step. All should have the same b
      __syncthreads();
    }
  }

  // perform exclusive prefix sum (scan) on binHistogram to get starting
  //location for each bin
  __global__ void prefix_sum(unsigned int * d_pos, unsigned int* d_in, int size) {


  }

  /*
  Radix sort implementation. Inspired from the provided reference function
  For each bit position, partition elts so that all elts with a 0 preceed those
  with a 1. When all bits have been processed the array is sorted.
  */
  void your_sort(unsigned int* const d_inputVals,
    unsigned int* const d_inputPos,
    unsigned int* const d_outputVals,
    unsigned int* const d_outputPos,
    const size_t numElems) {

      /*
      1) Histogram of the number of occurrences of each digit
      2) Exclusive Prefix Sum of Histogram
      3) Determine relative offset of each digit
      For example [0 0 1 1 0 0 1]
      ->  [0 1 0 1 2 3 2]
      4) Combine the results of steps 2 & 3 to determine the final
      output location for each element and move it there
      */
      const int numBits = 1;
      // numBins will always be 2 because we are counting 1's and 0's
      const int numBins = 1 << numBits;

      // set up vars
      unsigned int *d_binHistogram;
      unsigned int *d_binScan;
      unsigned int *vals_src = d_inputVals;
      unsigned int *pos_src = d_inputPos;
      unsigned int BIN_BYTES = numBins * sizeof(int);

      int threads = 1024;
      int blocks = numElems/threads;

      // allocate mem
      checkCudaErrors(hipMalloc((void **) &d_binHistogram, BIN_BYTES));
      checkCudaErrors(hipMalloc((void **) &d_binScan, BIN_BYTES));
      hipMemset(d_binHistogram, 0, BIN_BYTES);
      hipMemset(d_binScan, 0, BIN_BYTES);


      // zero out bins at each step
      hipMemset(d_binHistogram, 0, BIN_BYTES);
      hipMemset(d_binScan, 0, BIN_BYTES);

      //perform histogram of data & mask into bins
      bin_hist<<<blocks, threads>>>(d_binHistogram, d_inputVals, numElems, numBins);

      // copy back
      memcpy(d_outputPos, d_inputPos, BIN_BYTES);
      memcpy(d_outputVals, d_inputVals, BIN_BYTES);

      // Free allocated memory
      hipFree(d_binHistogram);
      hipFree(d_binScan);
}
