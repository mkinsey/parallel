#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green, and Blue is in it.
//The 'A' stands for Alpha and is used for transparency; it will be
//ignored in this homework.

//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{

  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A

  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;

  // calculate indexes
  int xi = threadIdx.x + (blockIdx.x * blockDim.x);
  int yi = threadIdx.y + (blockIdx.y * blockDim.y);
  int index = yi * numCols + xi;

  // apply formula
  greyImage[index] = .299f * rgbaImage[index].x + .587f * rgbaImage[index].y + .114f * rgbaImage[index].z;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{

  // Threads per block, 484
  const dim3 b_threads(22, 22);

  // Number of blocks
  const dim3 blocks(numCols/b_threads.x + 1, numRows/b_threads.y + 1);

  // create kernels
  rgba_to_greyscale<<<blocks, b_threads>>>(d_rgbaImage, d_greyImage, numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
